#include "hip/hip_runtime.h"
#include "kernels.h"
#include "hipblas.h"


void W_mult_H(real *WH, real *W, real *Htras, int N, int M, int K)
{
	cublasRgemm( 'T', 'n', 
		M,				/* [m] */ 
		N,				/* [n] */  
		K,				/* [k] */ 
		1,				/* alfa */ 
		Htras, K,			/* A[m][k], num columnas (lda) */ 
		W, K,				/* B[k][n], num columnas (ldb) */
		0,				/* beta */
		WH, M				/* C[m][n], num columnas (ldc) */
	);
}

__global__ void V_div_WH_device( real* V, real* WH, int ny, int nx )
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	int id = idy*nx+idx;

	// Make sure we do not go out of bounds
	if (idx<nx && idy<ny)
		WH[id] = V[id]/WH[id];
}

void V_div_WH( real* V, real* WH, int ny, int nx )
{
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	int a=nx/BLOCK_SIZE; if (nx % BLOCK_SIZE > 0) a++;
	int b=ny/BLOCK_SIZE; if (ny % BLOCK_SIZE > 0) b++;
	dim3 dimGrid(a,b);

	V_div_WH_device<<<dimGrid, dimBlock>>>( V, WH, ny, nx );


}

__global__ void init_accum_device( real *acc, real *X, int n, int nx )
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int i;

	// Make sure we do not go out of bounds
	if (idx<nx){
		acc[idx] = 0.0;
		for (i=0; i<n; i++)
			acc[idx] += X[i*nx+idx];
	}
}

void accum( real *acc, real* X, int n, int nx )
{

	int i;

	dim3 dimBlock(BLOCK_SIZE);
	int a=nx/BLOCK_SIZE; if (nx % BLOCK_SIZE > 0) a++;
	dim3 dimGrid(a);

	/* Init acc with 0s */
#if 1
	init_accum_device<<<dimGrid, dimBlock>>>( acc, X, n, nx );

#else
	cublasRaxpy(nx,		/* n, num de elementos del vector*/
		0.0,		/* alpha*/
		acc, 1,		/* vector x, incx */
		acc, 0		/* vector y, incy */ 		
	);
	for (i=0; i<n; i++)
		cublasRaxpy(nx,		/* n, num de elementos del vector*/
			1,		/* alpha*/
			X+i*nx, 1,	/* vector x, incx */
			acc, 1		/* vector y, incy */ 		
		);

#endif
}

void Wt_mult_WH( real *Haux, real *W, real *WH, int N, int M, int K)
{
	cublasRgemm( 'n', 'T', 
		K,				/* [m] */ 
		M,				/* [n] */  
		N,				/* [k] */ 
		1,				/* alfa */ 
		W, K,				/* A[m][k], num columnas (lda) */ 
		WH, M,				/* B[k][n], num columnas (ldb) */
		0,				/* beta */
		Haux, K				/* C[m][n], num columnas (ldc) */
	);
}

void WH_mult_Ht( real *Waux, real *WH, real *Htras, int N, int M, int K)
{
	cublasRgemm( 'n', 'n', 
		K,				/* [m] */ 
		N,				/* [n] */  
		M,				/* [k] */ 
		1,				/* alfa */ 
		Htras, K,			/* A[m][k], num columnas (lda) */ 
		WH, M,				/* B[k][n], num columnas (ldb) */
		0,				/* beta */
		Waux, K				/* C[m][n], num columnas (ldc) */
	);
}

__global__ void mult_M_div_vect_device(real *M, real *Maux, real *acc, int ny, int nx)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	int id = idy*nx+idx;

	// Make sure we do not go out of bounds
	if (idx<nx && idy<ny)
		M[id] = M[id]*Maux[id]/acc[idx];
}

void mult_M_div_vect(real *M, real *Maux, real *acc, int ny, int nx)
{
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	int a=nx/BLOCK_SIZE; if (nx % BLOCK_SIZE > 0) a++;
	int b=ny/BLOCK_SIZE; if (ny % BLOCK_SIZE > 0) b++;
	dim3 dimGrid(a,b);

	mult_M_div_vect_device<<<dimGrid, dimBlock>>>( M, Maux, acc, ny, nx );
}


__global__ void adjust_WH_device(real *M, int ny, int nx)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	int id = idy*nx+idx;

	// Make sure we do not go out of bounds
	if (idx<nx && idy<ny)
		if (M[id]<EPS)
			M[id] = EPS;
}

void adjust_WH_GPU(real *W, real *Htras, int N, int M, int K)
{
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	int a=K/BLOCK_SIZE;  if (K % BLOCK_SIZE > 0) a++;
	int bW=N/BLOCK_SIZE; if (N % BLOCK_SIZE > 0) bW++;
	int bH=M/BLOCK_SIZE; if (M % BLOCK_SIZE > 0) bH++;
	dim3 dimGridW(a,bW);
	dim3 dimGridH(a,bH);

	adjust_WH_device<<<dimGridW, dimBlock>>>( W, N, K );
	adjust_WH_device<<<dimGridH, dimBlock>>>( Htras, M, K );
}


